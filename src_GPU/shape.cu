#include "shape.h"
#include <hipblas.h>

Shape createSphere() {
	return { SHAPES_SPHERE, { 0, 0, 0, 1 }, createIdentityMatrix(4) };
}

__device__
int intersectCount(Shape shape, Ray ray, float* modelMatrixData) {
    hipblasHandle_t handle;

    hipblasCreate(&handle);
    hipblasDestroy(handle);

	int intersectionCount = 0;
	
	return intersectionCount;
}