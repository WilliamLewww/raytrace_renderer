#include "hip/hip_runtime.h"
#include "world.h"
#include <stdio.h>

World createDefaultWorld() {
	World world;

	world.pointLightCount = 1;
	world.shapeCount = 2;

	world.lightArray = new PointLight[1];
	world.lightArray[0] = {{ -10, 10, -10, 1 }, { 1, 1, 1, 1 }};

	world.shapeArray = new Shape[2];
	world.shapeArray[0] = createSphere();
	world.shapeArray[0].modelMatrix = createIdentityMatrix(4);
	world.shapeArray[0].inverseModelMatrix = inverse(world.shapeArray[0].modelMatrix);

	world.shapeArray[1] = createSphere();
	world.shapeArray[1].modelMatrix = createIdentityMatrix(4);
	world.shapeArray[1].inverseModelMatrix = inverse(world.shapeArray[1].modelMatrix);

	return world;
}

__device__
int intersectWorldCount(Shape* shapes, int shapeCount, Ray ray, float* allInverseModelMatrixData) {
	int intersectionCount = 0;

	for (int x = 0; x < shapeCount; x++) {
		float* inverseModelMatrixData = new float[16];
		memcpy(inverseModelMatrixData, &allInverseModelMatrixData[x * 16], 16 * sizeof(float));

		intersectionCount += intersectCount(shapes[x], ray, inverseModelMatrixData);
	}

	return intersectionCount;
}

__global__
void colorAtKernel(Tuple* colorBuffer, Ray* rays, int rayCount, Shape* shapes, int shapeCount, float* allInverseModelMatrixData) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int x = index; x < rayCount; x += stride) {
		int intersectionCount = intersectWorldCount(shapes, shapeCount, rays[x], allInverseModelMatrixData);

		if (intersectionCount > 0) {
			colorBuffer[x] = { 1, 0, 0, 1 };
		}
		else {
			colorBuffer[x] = { 0, 0, 0, 1 };
		}
	}
}

void colorAt(Tuple* colorOut, World world, Ray* rays, int rayCount) {
	Tuple* colorBuffer;
	Ray* rayBuffer;
	Shape* shapeBuffer;
	float* allInverseModelMatrixData;

	hipMallocManaged(&colorBuffer, rayCount*sizeof(Tuple));
	hipMallocManaged(&shapeBuffer, world.shapeCount*sizeof(Shape));
	hipMemcpy(shapeBuffer, world.shapeArray, world.shapeCount*sizeof(Shape), hipMemcpyHostToDevice);

	hipMallocManaged(&rayBuffer, rayCount*sizeof(Ray));
	hipMemcpy(rayBuffer, rays, rayCount*sizeof(Ray), hipMemcpyHostToDevice);

	hipMallocManaged(&allInverseModelMatrixData, world.shapeCount*16*sizeof(float));

	for (int x = 0; x < world.shapeCount; x++) {
		hipMemcpy(&allInverseModelMatrixData[(x * 16)], world.shapeArray[x].inverseModelMatrix.data, 16*sizeof(float), hipMemcpyHostToDevice);
	}

	int blockSize = 256;
	int numBlocks = (rayCount + blockSize - 1) / blockSize;
	colorAtKernel<<<numBlocks, blockSize>>>(colorBuffer, rayBuffer, rayCount, shapeBuffer, world.shapeCount, allInverseModelMatrixData);

	hipDeviceSynchronize();
	hipMemcpy(colorOut, colorBuffer, rayCount*sizeof(Tuple), hipMemcpyDeviceToHost);

	hipFree(colorBuffer);
	hipFree(shapeBuffer);
	hipFree(allInverseModelMatrixData);

	hipDeviceReset();
}